#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>

// Global pointers for the two input matrices, the result, and the transposed B
int *matrix1 = NULL;
int *matrix2 = NULL;
int *result = NULL;

int num_blocks, num_threads, N;

 __global__ void populate_matrix(int *Md, int *Nd, int N)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = Row * N + Col;

    if((idx < N * N))
    {
        // Each thread gets its own curandState:
        //   curand_init(idx,    // 64-bit seed value (you choose)
        //               idx,     // sequence number = “offset” in random sequence
        //               0,       // offset within sequence
        //               &state); // pointer to our cuRAND state
        hiprandState state;
        hiprand_init(idx, idx, 0, &state);

        // Generate a random int in [0, UINT_MAX]:
        unsigned int r1 = hiprand(&state);
        unsigned int r2 = hiprand(&state);

        int min = 1, max = 20;

        Md[idx] = (r1 % (max - min + 1)) + min;

        Nd[idx] = (r2 % (max - min + 1)) + min;
    }
}

__global__ void MatrixMulKernel(int *Md, int *Nd, int *Pd, int N)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    int idx = Row * N + Col;
    if((idx < N * N))
    {
        int PValue = 0;
        for(int k = 0; k < N; k++)
        {
            int Md_element = Md[Row * N + k];
            int Nd_element = Nd[k * N + Col];

            PValue += (Md_element * Nd_element);
        }
        Pd[idx] = PValue;
    }
}

int main(int argc, char * argv[])
{
    N = strtol(argv[1], NULL, 10);
    num_blocks = strtol(argv[2], NULL, 10);
    num_threads = N * N;

    int num_threads_groups = (num_threads + 1023) / 1024; // Each block can contain a moximum of 1024 threads
    while(num_blocks < num_threads_groups)
    {
        num_blocks++;
    }

    while(num_blocks > num_threads_groups)
    {
        num_blocks--;
    }

    dim3 blocks(num_blocks, num_blocks);
    dim3 threads_per_block;
    if(num_threads <= 1024)
        threads_per_block = dim3(N, N);

    else
        threads_per_block = dim3(32, 32);


    matrix1 = (int*)malloc(N * N * sizeof(int));
    matrix2 = (int*)malloc(N * N * sizeof(int));
    result = (int*)malloc(N * N * sizeof(int));

    int *Md, *Nd, *Pd;
    int size = N * N * sizeof(int);
    
    hipMalloc((void **) &Md, size);
    hipMalloc((void **) &Nd, size);
    hipMalloc((void **) &Pd, size);

    populate_matrix<<<blocks, threads_per_block>>>(Md, Nd, N);
    hipDeviceSynchronize();

    MatrixMulKernel<<<blocks, threads_per_block>>>(Md, Nd, Pd, N);
    hipDeviceSynchronize();

    hipMemcpy(result, Pd, size, hipMemcpyDeviceToHost);
    hipMemcpy(matrix1, Md, size, hipMemcpyDeviceToHost);
    hipMemcpy(matrix2, Nd, size, hipMemcpyDeviceToHost);

    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);

    printf("Matrix A is \n");
    for (int i = 0; i < N * N ; i++)
    {
        if(i % N == 0 && i != 0)
            printf("\n");
        printf("%d ", matrix1[i]);
    }

    printf("\n\n");

    // Print matrix B
    printf("Matrix B is \n");
    for (int i = 0; i < N * N ; i++)
    {
        if(i % N == 0 && i != 0)
            printf("\n");
        printf("%d ", matrix2[i]);
    }

    // Print the final product
    printf("\n\nMatrix A x B:\n");
    if (N <= 5)
        for (int i = 0; i < N * N; i++)
        {
            if (i % N == 0) printf("\n");
                printf("%d ", result[i]);
        }
        
    else
    {
        printf("Matrix size is greater than 5 X 5, so we will print only first 5 X 5 elements");
        for (int i = 0; i < 5; i++)
        {
            printf("\n");
            for (int j = 0; j < 5; j++)
            {
                printf("%d ", result[i * N + j]);  // print out only first 5 elements
            }
        }
    }

    printf("\n");
    free(matrix1);
    free(matrix2);
    free(result);

    return(0);
}