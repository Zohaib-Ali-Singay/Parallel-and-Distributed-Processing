
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Global variables to hold the number of blocks and the number of terms (N)
int num_blocks;
int N;

// CUDA kernel to compute a partial sum for pi approximation using the Leibniz series
__global__ void compute_pi(int num_blocks, int N, float *sum)
{
    double factor;

    // Compute the global row and column indices for this thread
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    // Flatten 2D indices into a 1D index
    int idx = Row * N + Col;

    // Determine the sign of the term: +1 for even idx, -1 for odd idx
    if (idx % 2 == 0)
        factor = 1.0;
    else
        factor = -1.0;

    // Each thread computes one term of the series if within bounds
    if (idx < N)
    {
        // Leibniz term: factor / (2*idx + 1)
        float term = factor / (2 * idx + 1);
        // Atomically add the term to the shared sum variable
        atomicAdd(sum, term);
    }
}

int main(int argc, char *argv[])
{
    // Host variable to store the sum of series
    float sum = 0.0f;

    // Parse command-line arguments: N (number of terms) and num_blocks (desired blocks)
    N = strtol(argv[1], NULL, 10);
    num_blocks = strtol(argv[2], NULL, 10);

    // Calculate the number of thread groups needed (each block can have up to 1024 threads)
    int num_threads_groups = (N + 1023) / 1024;

    // Adjust num_blocks to match the number of thread groups
    while (num_blocks < num_threads_groups)
    {
        num_blocks++;
    }
    while (num_blocks > num_threads_groups)
    {
        num_blocks--;
    }

    // Define the block and thread dimensions for kernel launch
    dim3 blocks(num_blocks);
    dim3 threads_per_block;

    // If N is small, use a square root decomposition for threads per block
    if (N <= 1024)
        threads_per_block = dim3((int)sqrt(N), (int)sqrt(N));
    else
        // Otherwise, use a 32x32 block of threads
        threads_per_block = dim3(32, 32);

    // Allocate device memory for the sum
    float *d_sum;
    hipMalloc((void**)&d_sum, sizeof(float));
    
    // Initialize device sum to zero
    hipMemset(d_sum, 0, sizeof(float));

    // Launch the kernel to compute the series
    compute_pi<<<blocks, threads_per_block>>>(num_blocks, N, d_sum);

    // Copy the computed sum back to host
    hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    // Multiply by 4 to get the approximation for pi and print the result
    printf("Value of pi is %f\n", 4.0 * sum);

    return 0;
}
